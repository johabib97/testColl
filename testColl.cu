#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#define ROOT 0
#define MAXNGPU 3
#define MAX_THREAD 512
#define M 200

#define BASE 5
#define POWER 13

//nvcc -I/usr/local/openmpi-4.1.4/include -L/usr/local/openmpi-4.1.4/lib -lmpi testColl.cu -o testC
// mpirun -np 1 testC 500 400 2

void cudaErrorCheck(hipError_t error, const char * msg){
   	 if ( error != hipSuccess){
   	 fprintf(stderr, "%s:%s\n ", msg, hipGetErrorString(error));
   	 exit(EXIT_FAILURE);}}

__host__ __device__ uint32_t abcFunct(uint32_t ua, uint32_t ub, uint32_t uc){
    	int a=ua;
    	int b=ub;
    	int c=uc;
    	uint32_t F=c+pow(a-b,2);
    	return F;
}


__host__ __device__ uint32_t baseFunct(uint32_t x, int r){
    r=r%(POWER+1);
    	uint32_t *arrayEquivX=(uint32_t*)malloc(sizeof(int)*(POWER+r));
    	uint32_t exp= pow(BASE, POWER);
    	uint32_t remnant;
    	uint32_t y;

    	remnant=x;

    	for (int i=0; i<r; i++) arrayEquivX[POWER+i]=0;

    	for (int i=0; i<POWER; i++){
            	exp=exp/BASE;
            	arrayEquivX[POWER-i-1]=remnant/exp;
            	remnant=remnant%exp;
            	//printf("in pos %d (exp %d) there is %d with remainder %d \n", POWER-1-i, exp, arrayEquivX[POWER-i-1],remnant);
    	}

    	y=0;

    	for (int i=0; i<POWER; i++){
            	//arrayEquivX[i]= abcFunct(arrayEquivX[i], arrayEquivX[i+1], arrayEquivX[i+j]);
            	//printf("(using %d) in pos %d there is %d \n", arrayEquivX[i+r], i, arrayEquivX[i]);
            	y=y+exp*abcFunct(arrayEquivX[i], arrayEquivX[i+1], arrayEquivX[i+r]);
            	exp=exp*BASE;
    	}
    	//printf("modulo is %d \n", exp);
    	y=y%exp;
    	return y;
}


    
int main (int argc, char** argv) {

//input validation
if(argc != 4){
    fprintf(stderr,"wrong number of inputs\n");
    return EXIT_FAILURE;}

uint32_t a=atoi(argv[1]);

if(a <=0){
   	 fprintf(stderr,"[ERROR] - lg must be > 0\n");
   	 return EXIT_FAILURE;}

//uint32_t ua=a;

uint32_t b=atoi(argv[1]);

if(b <=0){
    	fprintf(stderr,"[ERROR] - lg must be > 0\n");
    	return EXIT_FAILURE;}

int r=atoi(argv[3]);

uint32_t FA=baseFunct(a,r);
printf("F of %d is %d\n", a, FA);

uint32_t FB=baseFunct(b,r);
printf("F of %d is %d\n", b, FB);

if (FA==FB) printf("Collision confirmed on %d \n", FA);

}
